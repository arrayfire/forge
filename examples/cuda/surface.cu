#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#define USE_FORGE_CUDA_COPY_HELPERS
#include <ComputeCopy.h>
#include <cstdio>
#include <iostream>

const float XMIN = -8.0f;
const float XMAX = 8.f;
const float YMIN = -8.0f;
const float YMAX = 8.f;

const float DX = 0.5;
const size_t XSIZE = (size_t)((XMAX-XMIN)/DX);
const size_t YSIZE = (size_t)((YMAX-YMIN)/DX);

void kernel(float dx, float* dev_out);

int main(void)
{
    float *dev_out;

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other forge::* object to be created successfully
     */
    forge::Window wnd(1024, 768, "3d Surface Demo");
    wnd.makeCurrent();

    forge::Chart chart(FG_CHART_3D);
    chart.setAxesLimits(-10.f, 10.f, -10.f, 10.f, -0.5f, 1.f);
    chart.setAxesTitles("x-axis", "y-axis", "z-axis");

    forge::Surface surf = chart.surface(XSIZE, YSIZE, forge::f32);
    surf.setColor(FG_YELLOW);

    FORGE_CUDA_CHECK(hipMalloc((void**)&dev_out, XSIZE * YSIZE * 3 * sizeof(float) ));
    kernel(DX, dev_out);

    GfxHandle* handle;
    createGLBuffer(&handle, surf.vertices(), FORGE_VERTEX_BUFFER);
    /* copy your data into the vertex buffer object exposed by
     * forge::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, Forge provides copy headers
     * along with the library to help with this task
     */
    copyToGLBuffer(handle, (ComputeResourceHandle)dev_out, surf.verticesSize());

    do {
        wnd.draw(chart);
    } while(!wnd.close());

    FORGE_CUDA_CHECK(hipFree(dev_out));
    releaseGLBuffer(handle);
    return 0;
}


    __global__
void sincos_surf(float dx, float* out,
        const float XMIN, const float YMIN,
        const size_t XSIZE, const size_t YSIZE)
{
    int i = blockIdx.x * blockDim.x  + threadIdx.x;
    int j = blockIdx.y * blockDim.y  + threadIdx.y;

    float x= XMIN + i*dx;
    float y= YMIN + j*dx;
    if (i<XSIZE && j<YSIZE) {
        int offset = j + i * YSIZE;
        out[ 3 * offset     ] = x;
        out[ 3 * offset + 1 ] = y;
        float z = sqrt(x*x+y*y) + 2.2204e-16;
        out[ 3 * offset + 2 ] = sinf(z)/z;
    }
}

    inline
int divup(int a, int b)
{
    return (a+b-1)/b;
}

void kernel(float dx, float* dev_out)
{
    static const dim3 threads(8, 8);
    dim3 blocks(divup(XSIZE, threads.x),
            divup(YSIZE, threads.y));

    sincos_surf<<< blocks, threads >>>(dx, dev_out, XMIN, YMIN, XSIZE, YSIZE);
}
