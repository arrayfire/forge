#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#define USE_FORGE_CUDA_COPY_HELPERS
#include <ComputeCopy.h>

const unsigned DIMX = 640;
const unsigned DIMY = 480;
const float MINIMUM = 1.0f;
const float MAXIMUM = 20.f;
const float STEP    = 2.0f;
const int NELEMS    = (int)((MAXIMUM-MINIMUM+1)/STEP);

void generateColors(float* colors);

void generatePoints(float* points, float* dirs);

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

int main(void)
{
    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other forge::* object to be created successfully
     */
    forge::Window wnd(DIMX, DIMY, "3D Vector Field Demo");
    wnd.makeCurrent();

    forge::Chart chart(FG_CHART_3D);
    chart.setAxesLimits(MINIMUM-1.0f, MAXIMUM,
                        MINIMUM-1.0f, MAXIMUM,
                        MINIMUM-1.0f, MAXIMUM);
    chart.setAxesTitles("x-axis", "y-axis", "z-axis");

    int numElems = NELEMS*NELEMS*NELEMS;
    forge::VectorField field = chart.vectorField(numElems, forge::f32);
    field.setColor(0.f, 1.f, 0.f, 1.f);

    float* points;
    float* colors;
    float* dirs;

    FORGE_CUDA_CHECK(hipMalloc((void**)&points, 3*numElems*sizeof(float)));
    FORGE_CUDA_CHECK(hipMalloc((void**)&colors, 3*numElems*sizeof(float)));
    FORGE_CUDA_CHECK(hipMalloc((void**)&dirs, 3*numElems*sizeof(float)));

    generatePoints(points, dirs);
    generateColors(colors);

    GfxHandle* handles[3];
    createGLBuffer(&handles[0], field.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[1], field.colors(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[2], field.directions(), FORGE_VERTEX_BUFFER);

    copyToGLBuffer(handles[0], (ComputeResourceHandle)points, field.verticesSize());
    copyToGLBuffer(handles[1], (ComputeResourceHandle)colors,   field.colorsSize());
    copyToGLBuffer(handles[2], (ComputeResourceHandle)dirs, field.directionsSize());

    do {
        wnd.draw(chart);
    } while(!wnd.close());

    releaseGLBuffer(handles[0]);
    releaseGLBuffer(handles[1]);
    releaseGLBuffer(handles[2]);

    FORGE_CUDA_CHECK(hipFree(points));
    FORGE_CUDA_CHECK(hipFree(colors));
    FORGE_CUDA_CHECK(hipFree(dirs));

    return 0;
}

__global__
void genColorsKernel(float* colors, int nelems)
{
    const float AF_BLUE[4] = {0.0588f , 0.1137f , 0.2745f , 1.0f};
    const float AF_ORANGE[4] = {0.8588f , 0.6137f , 0.0745f , 1.0f};

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i<nelems) {
        if (i%2==0) {
            colors[3*i+0] = AF_ORANGE[0];
            colors[3*i+1] = AF_ORANGE[1];
            colors[3*i+2] = AF_ORANGE[2];
        } else {
            colors[3*i+0] = AF_BLUE[0];
            colors[3*i+1] = AF_BLUE[1];
            colors[3*i+2] = AF_BLUE[2];
        }
    }
}

void generateColors(float* colors)
{
    const int numElems = NELEMS*NELEMS*NELEMS;
    static const dim3 threads(512);
    dim3 blocks(divup(numElems, threads.x));

    genColorsKernel<<<blocks, threads>>>(colors, numElems);
}

__global__
void pointGenKernel(float* points, float* dirs, int nBBS0, int nelems, float minimum, float step)
{
    int k = blockIdx.x / nBBS0;
    int i = blockDim.x * (blockIdx.x-k*nBBS0) + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i<nelems && j<nelems && k<nelems) {
        float x = minimum + i*step;
        float y = minimum + j*step;
        float z = minimum + k*step;

        int id = i + j * nelems + k * nelems*nelems;

        points[3*id+0] = x;
        points[3*id+1] = y;
        points[3*id+2] = z;

        dirs[3*id+0] = x-10.f;
        dirs[3*id+1] = y-10.f;
        dirs[3*id+2] = z-10.f;
    }
}

void generatePoints(float* points, float* dirs)
{
    static dim3 threads(8, 8);

    int blk_x = divup(NELEMS, threads.x);
    int blk_y = divup(NELEMS, threads.y);

    dim3 blocks(blk_x*NELEMS, blk_y);

    pointGenKernel<<<blocks, threads>>>(points, dirs, blk_x, NELEMS, MINIMUM, STEP);
}
