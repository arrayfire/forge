#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define USE_FORGE_CUDA_COPY_HELPERS
#include <ComputeCopy.h>
#include <cstdio>
#include <iostream>

const unsigned DIMX = 1000;
const unsigned DIMY = 800;

static const float  DX = 0.1f;
static const float  FRANGE_START = 0.f;
static const float  FRANGE_END = 2 * 3.141592f;
static const size_t DATA_SIZE = (size_t)((FRANGE_END - FRANGE_START) / DX);

hiprandState_t* state;

void kernel(float* dev_out, int functionCode,
            float* colors, float* alphas, float* radii);

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

__global__
void setupRandomKernel(hiprandState *states, unsigned long long seed)
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, tid, 0, &states[tid]);
}

int main(void)
{
    FORGE_CUDA_CHECK(hipMalloc((void **)&state, DATA_SIZE*sizeof(hiprandState_t)));
    setupRandomKernel <<< divup(DATA_SIZE,32), 32 >>> (state, 314567);

    float *cos_out;
    float *tan_out;
    float *colors_out;
    float *alphas_out;
    float *radii_out;

    FORGE_CUDA_CHECK(hipMalloc((void**)&cos_out, sizeof(float) * DATA_SIZE * 2));
    FORGE_CUDA_CHECK(hipMalloc((void**)&tan_out, sizeof(float) * DATA_SIZE * 2));
    FORGE_CUDA_CHECK(hipMalloc((void**)&colors_out, sizeof(float) * DATA_SIZE * 3));
    FORGE_CUDA_CHECK(hipMalloc((void**)&alphas_out, sizeof(float) * DATA_SIZE));
    FORGE_CUDA_CHECK(hipMalloc((void**)&radii_out, sizeof(float) * DATA_SIZE));

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other forge::* object to be created successfully
     */
    forge::Window wnd(DIMX, DIMY, "Bubble chart with Transparency Demo");
    wnd.makeCurrent();

    forge::Chart chart(FG_CHART_2D);
    chart.setAxesLimits(FRANGE_START, FRANGE_END, -1.0f, 1.0f);

    /* Create several plot objects which creates the necessary
     * vertex buffer objects to hold the different plot types
     */
    forge::Plot plt1 = chart.plot(DATA_SIZE, forge::f32, FG_PLOT_LINE, FG_MARKER_TRIANGLE);
    forge::Plot plt2 = chart.plot(DATA_SIZE, forge::f32, FG_PLOT_LINE, FG_MARKER_CIRCLE);

    /* Set plot colors */
    plt1.setColor(FG_RED);
    plt2.setColor(FG_GREEN);            //use a forge predefined color
    /* Set plot legends */
    plt1.setLegend("Cosine");
    plt2.setLegend("Tangent");
    /* set plot global marker size */
    plt1.setMarkerSize(20);
    /* copy your data into the opengl buffer object exposed by
     * forge::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, Forge provides copy headers
     * along with the library to help with this task
     */

    GfxHandle* handles[5];

    // create GL-CUDA interop buffers
    createGLBuffer(&handles[0], plt1.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[1], plt2.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[2], plt2.colors(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[3], plt2.alphas(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[4], plt2.radii(), FORGE_VERTEX_BUFFER);

    kernel(cos_out, 0, NULL, NULL, NULL);
    kernel(tan_out, 1, colors_out, alphas_out, radii_out);

    // copy the data from compute buffer to graphics buffer
    copyToGLBuffer(handles[0], (ComputeResourceHandle)cos_out, plt1.verticesSize());
    copyToGLBuffer(handles[1], (ComputeResourceHandle)tan_out, plt2.verticesSize());

    /* update color value for tan graph */
    copyToGLBuffer(handles[2], (ComputeResourceHandle)colors_out, plt2.colorsSize());
    /* update alpha values for tan graph */
    copyToGLBuffer(handles[3], (ComputeResourceHandle)alphas_out, plt2.alphasSize());
    /* update marker sizes for tan graph markers */
    copyToGLBuffer(handles[4], (ComputeResourceHandle)radii_out, plt2.radiiSize());

    do {
        wnd.draw(chart);
    } while(!wnd.close());

    // destroy GL-CUDA Interop buffer
    releaseGLBuffer(handles[0]);
    releaseGLBuffer(handles[1]);
    releaseGLBuffer(handles[2]);
    releaseGLBuffer(handles[3]);
    releaseGLBuffer(handles[4]);
    // destroy CUDA handles
    FORGE_CUDA_CHECK(hipFree(cos_out));
    FORGE_CUDA_CHECK(hipFree(tan_out));
    FORGE_CUDA_CHECK(hipFree(colors_out));
    FORGE_CUDA_CHECK(hipFree(alphas_out));
    FORGE_CUDA_CHECK(hipFree(radii_out));

    return 0;
}

__global__
void mapKernel(float* out, int functionCode, float frange_start, float dx)
{
    int id = blockIdx.x * blockDim.x  + threadIdx.x;
    float x = frange_start + id*dx;
    float y;

    switch(functionCode) {
        case 0: y = cos(x); break;
        case 1: y = tan(x); break;
        default: y = sin(x); break;
    }

    out[2*id+0] = x;
    out[2*id+1] = y;
}

__global__
void colorsKernel(float* colors, hiprandState *states)
{
    int id = blockIdx.x * blockDim.x  + threadIdx.x;

    colors[3*id+0] = hiprand_uniform(&states[id]);
    colors[3*id+1] = hiprand_uniform(&states[id]);
    colors[3*id+2] = hiprand_uniform(&states[id]);
}

__global__
void randKernel(float* out, hiprandState *states, float min, float scale)
{
    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    out[id] = hiprand_uniform(&states[id])*scale + min;
}

void kernel(float* dev_out, int functionCode,
            float* colors, float* alphas, float* radii)
{
    static const dim3 threads(32);
    dim3 blocks(divup(DATA_SIZE, 32));

    mapKernel<<< blocks, threads >>>(dev_out, functionCode, FRANGE_START, DX);

    if (colors)
        colorsKernel<<< blocks, threads >>>(colors, state);

    if (alphas)
        randKernel<<< blocks, threads >>>(alphas, state, 0, 1);

    if (radii)
        randKernel<<< blocks, threads >>>(radii, state, 20, 60);
}
