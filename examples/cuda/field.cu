#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#define USE_FORGE_CUDA_COPY_HELPERS
#include <ComputeCopy.h>

#define PI 3.14159265359

const unsigned DIMX = 640;
const unsigned DIMY = 480;
const float MINIMUM = 1.0f;
const float MAXIMUM = 20.f;
const float STEP    = 2.0f;
const float NELEMS  = (MAXIMUM-MINIMUM+1)/STEP;
const unsigned DPOINTS[] = {5, 5, 5, 15, 15, 5, 15, 15};

void generatePoints(float* points, float* dirs);

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

int main(void)
{
    unsigned* dpoints;
    float* points;
    float* dirs;
    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other forge::* object to be created successfully
     */
    forge::Window wnd(DIMX, DIMY, "Vector Field Demo");
    wnd.makeCurrent();

    forge::Chart chart(FG_CHART_2D);
    chart.setAxesLimits(MINIMUM-1.0f, MAXIMUM, MINIMUM-1.0f, MAXIMUM);
    chart.setAxesTitles("x-axis", "y-axis");

    forge::Plot divPoints = chart.plot(4, forge::u32, FG_PLOT_SCATTER, FG_MARKER_CIRCLE);
    divPoints.setColor(0.9f, 0.9f, 0.0f, 1.f);
    divPoints.setLegend("Convergence Points");
    divPoints.setMarkerSize(24);

    size_t npoints = (size_t)(NELEMS*NELEMS);

    forge::VectorField field = chart.vectorField((unsigned)(npoints), forge::f32);
    field.setColor(0.f, 0.6f, 0.3f, 1.f);

    FORGE_CUDA_CHECK(hipMalloc((void**)&dpoints, 8*sizeof(unsigned)));
    FORGE_CUDA_CHECK(hipMalloc((void**)&points, 2*npoints*sizeof(float)));
    FORGE_CUDA_CHECK(hipMalloc((void**)&dirs, 2*npoints*sizeof(float)));

    GfxHandle* handles[3];

    createGLBuffer(&handles[0], divPoints.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[1], field.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[2], field.directions(), FORGE_VERTEX_BUFFER);

    FORGE_CUDA_CHECK(hipMemcpy(dpoints, DPOINTS, 8*sizeof(unsigned), hipMemcpyHostToDevice));
    generatePoints(points, dirs);

    copyToGLBuffer(handles[0], (ComputeResourceHandle)dpoints, divPoints.verticesSize());

    copyToGLBuffer(handles[1], (ComputeResourceHandle)points, field.verticesSize());
    copyToGLBuffer(handles[2], (ComputeResourceHandle)dirs, field.directionsSize());

    do {
        wnd.draw(chart);
    } while(!wnd.close());

    // destroy GL-CUDA interop buffers
    releaseGLBuffer(handles[0]);
    releaseGLBuffer(handles[1]);
    releaseGLBuffer(handles[2]);
    // destroy CUDA handles
    FORGE_CUDA_CHECK(hipFree(dpoints));
    FORGE_CUDA_CHECK(hipFree(points));
    FORGE_CUDA_CHECK(hipFree(dirs));

    return 0;
}

__global__
void pointGenKernel(float* points, float* dirs, int nelems, float minimum, float step)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i<nelems && j<nelems) {
        int id  = i + j * nelems;

        float x = minimum + i*step;
        float y = minimum + j*step;

        points[2*id+0] = x;
        points[2*id+1] = y;

        dirs[2*id+0] = sinf(2.0f*PI*x/10.f);
        dirs[2*id+1] = sinf(2.0f*PI*y/10.f);
    }
}

void generatePoints(float* points, float* dirs)
{
    static dim3 threads(8, 8);
    dim3 blocks(divup((int)(NELEMS), threads.x),
                divup((int)(NELEMS), threads.y));

    pointGenKernel<<<blocks, threads>>>(points, dirs, (int)(NELEMS), MINIMUM, STEP);
}
