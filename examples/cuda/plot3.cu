#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#define USE_FORGE_CUDA_COPY_HELPERS
#include <ComputeCopy.h>
#include <cstdio>
#include <iostream>

const unsigned DIMX = 1000;
const unsigned DIMY = 800;

static const float ZMIN = 0.1f;
static const float ZMAX = 10.f;

const float DX = 0.005f;
const size_t ZSIZE = (size_t)((ZMAX-ZMIN)/DX+1);

void kernel(float t, float dx, float* dev_out);

int main(void)
{
    float *dev_out;

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other forge::* object to be created successfully
     */
    forge::Window wnd(DIMX, DIMY, "Three dimensional line plot demo");
    wnd.makeCurrent();

    forge::Chart chart(FG_CHART_3D);

    chart.setAxesLabelFormat("%3.1f", "%3.1f", "%.2e");

    chart.setAxesLimits(-1.1f, 1.1f, -1.1f, 1.1f, 0.f, 10.f);

    chart.setAxesTitles("x-axis", "y-axis", "z-axis");

    forge::Plot plot3 = chart.plot(ZSIZE, forge::f32);

    static float t=0;
    FORGE_CUDA_CHECK(hipMalloc((void**)&dev_out, ZSIZE * 3 * sizeof(float) ));
    kernel(t, DX, dev_out);

    GfxHandle* handle;
    createGLBuffer(&handle, plot3.vertices(), FORGE_VERTEX_BUFFER);

    /* copy your data into the vertex buffer object exposed by
     * forge::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, Forge provides copy headers
     * along with the library to help with this task
     */
    copyToGLBuffer(handle, (ComputeResourceHandle)dev_out, plot3.verticesSize());

    do {
        t+=0.01f;
        kernel(t, DX, dev_out);
        copyToGLBuffer(handle, (ComputeResourceHandle)dev_out, plot3.verticesSize());
        wnd.draw(chart);
    } while(!wnd.close());

    FORGE_CUDA_CHECK(hipFree(dev_out));
    releaseGLBuffer(handle);
    return 0;
}


__global__
void generateCurve(float t, float dx, float* out, const float ZMIN, const size_t ZSIZE)
{
    int offset = blockIdx.x * blockDim.x  + threadIdx.x;

    float z = ZMIN + offset*dx;
    if(offset < ZSIZE) {
        out[ 3 * offset     ] = cos(z*t+t)/z;
        out[ 3 * offset + 1 ] = sin(z*t+t)/z;
        out[ 3 * offset + 2 ] = z + 0.1*sin(t);
    }
}

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

void kernel(float t, float dx, float* dev_out)
{
    static const dim3 threads(1024);
    dim3 blocks(divup(ZSIZE, 1024));

    generateCurve<<< blocks, threads >>>(t, dx, dev_out, ZMIN, ZSIZE);
}
