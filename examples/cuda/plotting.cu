#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#define USE_FORGE_CUDA_COPY_HELPERS
#include <ComputeCopy.h>
#include <cstdio>
#include <iostream>

const unsigned DIMX = 1000;
const unsigned DIMY = 800;

static const float    dx = 0.1f;
static const float    FRANGE_START = 0.f;
static const float    FRANGE_END = 2 * 3.141592f;
static const size_t   DATA_SIZE = (size_t)(( FRANGE_END - FRANGE_START ) / dx);

void kernel(float* dev_out, int functionCode);

int main(void)
{
    float *sin_out;
    float *cos_out;
    float *tan_out;
    float *log_out;

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other forge::* object to be created successfully
     */
    forge::Window wnd(DIMX, DIMY, "Plotting Demo");
    wnd.makeCurrent();

    forge::Chart chart(FG_CHART_2D);
    chart.setAxesLimits(FRANGE_START, FRANGE_END, -1.0f, 1.0f);

    /* Create several plot objects which creates the necessary
     * vertex buffer objects to hold the different plot types
     */
    forge::Plot plt0 = chart.plot( DATA_SIZE, forge::f32);                                 //create a default plot
    forge::Plot plt1 = chart.plot( DATA_SIZE, forge::f32, FG_PLOT_LINE, FG_MARKER_NONE);       //or specify a specific plot type
    forge::Plot plt2 = chart.plot( DATA_SIZE, forge::f32, FG_PLOT_LINE, FG_MARKER_TRIANGLE);   //last parameter specifies marker shape
    forge::Plot plt3 = chart.plot( DATA_SIZE, forge::f32, FG_PLOT_SCATTER, FG_MARKER_CROSS);

    /*
     * Set plot colors
     */
    plt0.setColor(FG_RED);
    plt1.setColor(FG_BLUE);
    plt2.setColor(FG_YELLOW);            //use a forge predefined color
    plt3.setColor((forge::Color) 0x257973FF);  //or any hex-valued color
    /*
     * Set plot legends
     */
    plt0.setLegend("Sine");
    plt1.setLegend("Cosine");
    plt2.setLegend("Tangent");
    plt3.setLegend("Log base 10");

    FORGE_CUDA_CHECK(hipMalloc((void**)&sin_out, sizeof(float) * DATA_SIZE * 2));
    FORGE_CUDA_CHECK(hipMalloc((void**)&cos_out, sizeof(float) * DATA_SIZE * 2));
    FORGE_CUDA_CHECK(hipMalloc((void**)&tan_out, sizeof(float) * DATA_SIZE * 2));
    FORGE_CUDA_CHECK(hipMalloc((void**)&log_out, sizeof(float) * DATA_SIZE * 2));

    kernel(sin_out, 0);
    kernel(cos_out, 1);
    kernel(tan_out, 2);
    kernel(log_out, 3);

    GfxHandle* handles[4];
    createGLBuffer(&handles[0], plt0.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[1], plt1.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[2], plt2.vertices(), FORGE_VERTEX_BUFFER);
    createGLBuffer(&handles[3], plt3.vertices(), FORGE_VERTEX_BUFFER);

    /* copy your data into the vertex buffer object exposed by
     * forge::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, Forge provides copy headers
     * along with the library to help with this task
     */
    copyToGLBuffer(handles[0], (ComputeResourceHandle)sin_out, plt0.verticesSize());
    copyToGLBuffer(handles[1], (ComputeResourceHandle)cos_out, plt1.verticesSize());
    copyToGLBuffer(handles[2], (ComputeResourceHandle)tan_out, plt2.verticesSize());
    copyToGLBuffer(handles[3], (ComputeResourceHandle)log_out, plt3.verticesSize());

    do {
        wnd.draw(chart);
    } while(!wnd.close());

    FORGE_CUDA_CHECK(hipFree(sin_out));
    FORGE_CUDA_CHECK(hipFree(cos_out));
    FORGE_CUDA_CHECK(hipFree(tan_out));
    FORGE_CUDA_CHECK(hipFree(log_out));
    releaseGLBuffer(handles[0]);
    releaseGLBuffer(handles[1]);
    releaseGLBuffer(handles[2]);
    releaseGLBuffer(handles[3]);

    return 0;
}

__global__
void simple_sinf(float* out, const size_t _data_size, int fnCode, const float _dx, const float _frange_start)
{
    int i = blockIdx.x * blockDim.x  + threadIdx.x;

    if (i < _data_size) {
        float x  = _frange_start + i * _dx;
        int idx  = 2 * i;
        out[idx] = x;

        switch(fnCode) {
            case 0:
                out[ idx + 1 ] = sinf(x);
                break;
            case 1:
                out[ idx + 1 ] = cosf(x);
                break;
            case 2:
                out[ idx + 1 ] = tanf(x);
                break;
            case 3:
                out[ idx + 1 ] = log10f(x);
                break;
        }
    }
}

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

void kernel(float* dev_out, int functionCode)
{
    static const dim3 threads(1024);
    dim3 blocks(divup(DATA_SIZE, 1024));

    simple_sinf << < blocks, threads >> >(dev_out, DATA_SIZE, functionCode, dx, FRANGE_START);
}
